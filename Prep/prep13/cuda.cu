#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_insert(CudaPic picBG, CudaPic picFG, CudaPic res, uchar3 tint)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= picBG.m_size.x) return;
    if(y >= picBG.m_size.y) return;

    uchar4 l_background = picBG.getData<uchar4>(x, y);
    uchar4 l_foreground = picFG.getData<uchar4>(x, y);

    if(l_background.w == 0) {
        uchar3 l_color = make_uchar3(l_foreground.x, l_foreground.y, l_foreground.z);
        uchar3 l_tint = tint;

        uchar3 l_result = make_uchar3(
            (l_color.x * l_tint.x) / 255,
            (l_color.y * l_tint.y) / 255,
            (l_color.z * l_tint.z) / 255
        );

        res.setData<uchar4>(x, y, make_uchar4(l_result.x, l_result.y, l_result.z, 255));
    } else {
        res.setData<uchar4>(x, y, l_background);
    }
}

__global__ void kernel_upChannel(CudaPic inPic, CudaPic outPic)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    uchar3 l_color = inPic.getData<uchar3>(x, y);
    outPic.setData<uchar4>(x, y, make_uchar4(l_color.x, l_color.y, l_color.z, 255));
}

void cuda_insert(CudaPic picBG, CudaPic picFG, CudaPic res, uchar3 tint)
{
    hipError_t l_cerr;

    int block_size = 32;

    dim3 l_blocks((picBG.m_size.x + block_size - 1) / block_size, (picBG.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    kernel_insert<<<l_blocks, l_threads>>>(picBG, picFG, res, tint);

    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    hipDeviceSynchronize();
}

void cuda_upChannel(CudaPic inPic, CudaPic outPic)
{
    hipError_t l_cerr;

    int block_size = 32;

    dim3 l_blocks((inPic.m_size.x + block_size - 1) / block_size, (inPic.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    kernel_upChannel<<<l_blocks, l_threads>>>(inPic, outPic);

    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    hipDeviceSynchronize();
}
