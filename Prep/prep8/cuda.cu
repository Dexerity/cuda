#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Kernel function to mirror the image along the specified axis
__global__ void kernel_mirror(CudaPic inPic, CudaPic outPic, int axis)
{
    // Calculate the x and y coordinates of the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the image boundaries
    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    // Get the pixel value at (x, y)
    uchar3 l_bgra = inPic.getData<uchar3>(x, y);

    if(axis == 0)
    {
        // Mirror the pixel along the x-axis
        outPic.setData<uchar3>(x, inPic.m_size.y - y - 1, l_bgra);
    }
    else
    {
        // Mirror the pixel along the y-axis
        outPic.setData<uchar3>(inPic.m_size.x - x - 1, y, l_bgra);
    }
}

// Kernel function to darken the image based on the y-coordinate
__global__ void kernel_darken(CudaPic inPic, CudaPic outPic)
{
    // Calculate the x and y coordinates of the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the image boundaries
    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    // Get the pixel value at (x, y)
    uchar3 l_bgra = inPic.getData<uchar3>(x, y);

    // Darken the pixel based on the y-coordinate
    l_bgra = make_uchar3(l_bgra.x * (y / (float)inPic.m_size.y), l_bgra.y * (y / (float)inPic.m_size.y), l_bgra.z * (y / (float)inPic.m_size.y));

    // Set the darkened pixel in the output image
    outPic.setData<uchar3>(x, y, l_bgra);
}

// Kernel function to double the width of the image
__global__ void kernel_double(CudaPic inPic, CudaPic outPic)
{
    // Calculate the x and y coordinates of the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the image boundaries
    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    // Get the pixel value at (x, y)
    uchar3 l_bgra = inPic.getData<uchar3>(x, y);

    // Set the pixel in the output image
    outPic.setData<uchar3>(x, y, l_bgra);
    outPic.setData<uchar3>(x + inPic.m_size.x, y, l_bgra);
}

// Function to mirror the image using CUDA
void cuda_mirror(CudaPic inPic, CudaPic outPic, int axis)
{
    hipError_t l_cerr;

    int block_size = 32;

    // Calculate the number of blocks and threads per block
    dim3 l_blocks((inPic.m_size.x + block_size - 1) / block_size, (inPic.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    // Launch the mirror kernel
    kernel_mirror<<<l_blocks, l_threads>>>(inPic, outPic, axis);

    // Check for any CUDA errors
    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    // Synchronize the device
    hipDeviceSynchronize();
}

// Function to darken the image using CUDA
void cuda_darken(CudaPic inPic, CudaPic outPic)
{
    hipError_t l_cerr;

    int block_size = 32;

    // Calculate the number of blocks and threads per block
    dim3 l_blocks((inPic.m_size.x + block_size - 1) / block_size, (inPic.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    // Launch the darken kernel
    kernel_darken<<<l_blocks, l_threads>>>(inPic, outPic);

    // Check for any CUDA errors
    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    // Synchronize the device
    hipDeviceSynchronize();
}

// Function to double the width of the image using CUDA
Mat cuda_double(CudaPic inPic)
{
    hipError_t l_cerr;

    // Create an output matrix with double the width
    Mat outMat = Mat(inPic.m_size.y, inPic.m_size.x * 2, CV_8UC3);
    CudaPic outPic = CudaPic(outMat);

    int block_size = 32;

    // Calculate the number of blocks and threads per block
    dim3 l_blocks((inPic.m_size.x + block_size - 1) / block_size, (inPic.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    // Launch the double kernel
    kernel_double<<<l_blocks, l_threads>>>(inPic, outPic);

    // Check for any CUDA errors
    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    // Synchronize the device
    hipDeviceSynchronize();

    return outMat;
}