#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_img.h"


// Kernel to clear the image
__global__ void kernel_run_sim_step(CudaPic inPic, CudaPic outPic, int ruleString1, int ruleString2)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is outside the image boundaries
    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    int bRule[8] = {0, 0, 0, 0, 0, 0, 0, 0};
    int sRule[8] = {0, 0, 0, 0, 0, 0, 0, 0};

    for(int i = 0; i < 8; i++)
    {
        
        bRule[i] = (int)(ruleString1 / powf(10, i)) % 10;
        sRule[i] = (int)(ruleString2 / powf(10, i)) % 10;
    }

    uchar1 tmp = inPic.getData<uchar1>(x, y);

    int alive = 0;
    int dead = 0;

    for (int i = -1; i <= 1; i++)
    {
        for (int j = -1; j <= 1; j++)
        {
            if (i == 0 && j == 0)
                continue;

            uchar1 tmp2;

            if(x + i < 0 || x + i >= inPic.m_size.x)
                tmp2 = make_uchar1(0);
            else if(y + j < 0 || y + j >= inPic.m_size.y)
                tmp2 = make_uchar1(0);
            else
                tmp2 = inPic.getData<uchar1>(x + i, y + j);

            if (tmp2.x == 255)
            {
                alive++;
            }
            else
            {
                dead++;
            }
        }
    }

    if(tmp.x == 255)
    {
        for(int i = 0; i < 8; i++)
        {
            if(sRule[i] == 0)
                continue;

            if(alive == sRule[i])
            {
                outPic.setData<uchar1>(x, y, make_uchar1(255));
                break;
            }
                outPic.setData<uchar1>(x, y, make_uchar1(0));
        }

        if(ruleString2 == 0)
            outPic.setData<uchar1>(x, y, make_uchar1(0));
    }
    else if(tmp.x == 0)
    {
        for(int i = 0; i < 8; i++)
        {
            if(bRule[i] == 0)
                continue;

            if(alive == bRule[i])
            {
                outPic.setData<uchar1>(x, y, make_uchar1(255));
                break;
            }
                outPic.setData<uchar1>(x, y, make_uchar1(0));
        }

        if(ruleString1 == 0)
            outPic.setData<uchar1>(x, y, make_uchar1(0));
    }
    else
    {
        outPic.setData<uchar1>(x, y, make_uchar1(0));
    }

}

__global__ void kernel_random_line(CudaPic inPic)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is outside the image boundaries
    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    if(y != 1) return;

    hiprandState state;
    hiprand_init(0, x, 0, &state);
    

    uchar l_new_pixel = hiprand(&state) % 2 * 255;
    inPic.setData<uchar1>(x, y, make_uchar1(l_new_pixel));
}

__global__ void kernel_random_canvas(CudaPic pic)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is outside the image boundaries
    if(x >= pic.m_size.x) return;
    if(y >= pic.m_size.y) return;

    // Generate a random number between 0 and 1
    hiprandState state;
    hiprand_init(0, x, 0, &state);
    float random = hiprand_uniform(&state);

    if(random < 0.5)
    {
        pic.setData<uchar1>(x, y, make_uchar1(255));
    }
    else
    {
        pic.setData<uchar1>(x, y, make_uchar1(0));
    }
}

void cuda_run_sim_step(CudaPic inPic, CudaPic outPic, int ruleString, int ruleString2)
{
    hipError_t l_cuda_err;

    // Set the block size
    int l_block_size = 32;

    // Calculate the number of blocks and threads
    dim3 l_blocks((inPic.m_size.x + l_block_size - 1) / l_block_size,
                  (inPic.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_run_sim_step<<<l_blocks, l_threads>>>(inPic, outPic, ruleString, ruleString2);

    if((l_cuda_err = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cuda_err));
    }

    // Synchronize the device
    hipDeviceSynchronize();
}

void cuda_random_canvas(CudaPic pic)
{
    hipError_t l_cuda_err;

    // Set the block size
    int l_block_size = 32;

    // Calculate the number of blocks and threads
    dim3 l_blocks((pic.m_size.x + l_block_size - 1) / l_block_size,
                  (pic.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_random_canvas<<<l_blocks, l_threads>>>(pic);

    if((l_cuda_err = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cuda_err));
    }

    // Synchronize the device
    hipDeviceSynchronize();
}

void cuda_random_line(CudaPic inPic)
{
    hipError_t l_cuda_err;

    // Set the block size
    int l_block_size = 32;

    // Calculate the number of blocks and threads
    dim3 l_blocks((inPic.m_size.x + l_block_size - 1) / l_block_size,
                  (inPic.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_random_line<<<l_blocks, l_threads>>>(inPic);

    if((l_cuda_err = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cuda_err));
    }

    // Synchronize the device
    hipDeviceSynchronize();
}

void cuda_create_stable(CudaPic pic, int x, int y)
{
    pic.setData<uchar1>(x, y, make_uchar1(255));
    pic.setData<uchar1>(x + 1, y, make_uchar1(255));
    pic.setData<uchar1>(x, y + 1, make_uchar1(255));
    pic.setData<uchar1>(x + 1, y + 1, make_uchar1(255));
}

void cuda_create_glider(CudaPic pic, int x, int y)
{
    pic.setData<uchar1>(x, y, make_uchar1(255));
    pic.setData<uchar1>(x + 1, y + 1, make_uchar1(255));
    pic.setData<uchar1>(x + 2, y + 1, make_uchar1(255));
    pic.setData<uchar1>(x + 2, y, make_uchar1(255));
    pic.setData<uchar1>(x + 2, y - 1, make_uchar1(255));
}
