#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_mult(CudaPic inPic, CudaPic outPic)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    int half_x = inPic.m_size.x / 2;
    int half_y = inPic.m_size.y / 2;

    uchar4 l_color = inPic.getData<uchar4>(x, y);

    outPic.setData<uchar4>(x / 2, y / 2, l_color);
    outPic.setData<uchar4>(x / 2 + half_x, y / 2, l_color);
    outPic.setData<uchar4>(x / 2, y / 2 + half_y, l_color);
    outPic.setData<uchar4>(x / 2 + half_x, y / 2 + half_y, l_color);
}

__global__ void kernel_insertAt(CudaPic picBG, CudaPic picFG, CudaPic res, int xP, int yP)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= picBG.m_size.x) return;
    if(y >= picBG.m_size.y) return;

    res.setData<uchar4>(x, y, picBG.getData<uchar4>(x, y));

    //insert image at x, y, even if half of it is outside
    if(x >= xP && y >= yP && x < xP + picFG.m_size.x && y < yP + picFG.m_size.y)
    {
        uchar4 l_color = picFG.getData<uchar4>(x - xP, y - yP);
        if(l_color.w == 0) return;
        res.setData<uchar4>(x, y, l_color);
    }
}

__global__ void kernel_upChannel(CudaPic inPic, CudaPic outPic)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= inPic.m_size.x) return;
    if(y >= inPic.m_size.y) return;

    uchar3 l_color = inPic.getData<uchar3>(x, y);
    outPic.setData<uchar4>(x, y, make_uchar4(l_color.x, l_color.y, l_color.z, 255));
}

void cuda_mult(CudaPic inPic, CudaPic outPic)
{
    hipError_t l_cerr;

    int block_size = 32;

    dim3 l_blocks((inPic.m_size.x + block_size - 1) / block_size, (inPic.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    kernel_mult<<<l_blocks, l_threads>>>(inPic, outPic);

    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    hipDeviceSynchronize();
}

void cuda_insertAt(CudaPic picBG, CudaPic picFG, CudaPic res, int xP, int yP)
{
    hipError_t l_cerr;

    int block_size = 32;

    dim3 l_blocks((picBG.m_size.x + block_size - 1) / block_size, (picBG.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    kernel_insertAt<<<l_blocks, l_threads>>>(picBG, picFG, res, xP, yP);

    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    hipDeviceSynchronize();
}

void cuda_upChannel(CudaPic inPic, CudaPic outPic)
{
    hipError_t l_cerr;

    int block_size = 32;

    dim3 l_blocks((inPic.m_size.x + block_size - 1) / block_size, (inPic.m_size.y + block_size - 1) / block_size);
    dim3 l_threads(block_size, block_size);

    kernel_upChannel<<<l_blocks, l_threads>>>(inPic, outPic);

    if((l_cerr = hipGetLastError()) != hipSuccess) {
        printf("CUDA error [%d]: %s\n", __LINE__, hipGetErrorString(l_cerr));
    }

    hipDeviceSynchronize();
}
